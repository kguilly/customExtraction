#include "hip/hip_runtime.h"
#include "decompress_cuda.h"
#include "iostream"
#include <stdlib.h>

/*
Funtion that gets device information:
 - number of available GPUs
 - max number of threads per block
*/

deviceInfo_t* available_gpus;

void device_information (int num_devices) {
    
    // int num_devices;
    // hipGetDeviceCount(&num_devices);
    // available_gpus = new DeviceInfo[num_devices];
    // make sure the array is allocated before calling the function
    
    for (int i=0; i<num_devices; i++) {
        hipSetDevice(i);
        hipDeviceProp_t gpu;
        hipGetDeviceProperties(&gpu, 0);
        DeviceInfo dev;
        dev.name = gpu.name;
        dev.clockRate = gpu.clockRate;
        dev.maxThreadsperBlock = gpu.maxThreadsPerBlock;
        dev.maxBlocksperMulti = gpu.maxBlocksPerMultiProcessor;

        available_gpus[i] = dev;
    }
}

/*
Function that performs the index extraction
*/
__global__ void find_nearest_points (station_t * stationArr, float * lats, float * lons, int numStations, int num_lons_lats) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < numStations) {
        float min_distance = 999;
        int min_index = -1;
        
        station_t * curr_station = &stationArr[id];
        float st_lat = (curr_station->latll + curr_station->latur) / 2.0;
        float st_lon = (curr_station->lonll + curr_station->lonur) / 2.0;

        for (int i=0; i<numberOfPoints; i++) {
            float lat = lats[i];
            float lon = lons[i]; 
            float distance = sqrt(pow((st_lat - lat), 2) + pow((st_lon - lon), 2));

            if (distance < min_distance) {
                min_distance = distance;
                min_index = i;
            }
        }

        curr_station->closestPoint = min_index;
    }
    
}

/*
Function that orchestrates the index extraction
*/
void index_extraction (station_t & stationArr, float* lats, float* lons, int numStations, int numberOfPoints) { 
    
    // get useful device properties
    int num_devices;
    hipGetDeviceCount(&num_devices);
    available_gpus = new DeviceInfo[num_devices];
    hipSetDevice(0);
    int max_threads = available_gpus[0].maxThreadsPerBlock;
    int max_blocks = available_gpus[0].maxBlocksPerMulti;

    int num_threads_to_use = max_threads / 2;
    int num_blocks_to_use = max_blocks / 2;

    // make device copies of host params
    station_t* d_stationArr;
    float* d_lats, d_lons;
    

    // allocate each of the arguments to the GPU
    if (hipMalloc(&d_stationArr, sizeof(station_t) * numStations) != hipSuccess) {
        std::cout << "stationArr could not be allocated to GPU" << std::endl;
        delete[] available_gpus;
        return;
    }
    if (hipMalloc(&d_lats, sizeof(float) * numberOfPoints) != hipSuccess) {
        std::cout << "Grib lats could not be allocated to the GPU" << std::endl;
        hipFree(d_stationArr);
        delete[] available_gpus;
        return;
    }
    if (hipMalloc(&d_lons, sizeof(float) * numberOfPoints) != hipSuccess) {
        std::cout << "Grib lons could not be allocated to the GPU" << std::endl;
        hipFree(d_stationArr);
        hipFree(d_lats);
        delete[] available_gpus;
    }

    // copy each of the arguments over to the gpu
    if (hipMemcpy(d_stationArr, stationArr, sizeof(station_t) * numStations, hipMemcpyHostToDevice) != hipSuccess) {
        std::cout << "The stationArr could not be copied to the GPU" << std::endl;
        hipFree(d_stationArr);
        hipFree(d_lats);
        hipFree(d_lons);
        delete[] available_gpus;
        return;
    }
    if (hipMemcpy(d_lats, lats, sizeof(float) * numberOfPoints, hipMemcpyHostToDevice) != hipSuccess) {
        std::cout << "The grib lats could not be copied to the GPU" << std::endl;
        hipFree(d_stationArr);
        hipFree(d_lats);
        hipFree(d_lons);
        delete[] available_gpus;
        return;
    }
    if (hipMemcpy(d_lons, lons, sizeof(float) * numberOfPoints, hipMemcpyHostToDevice) != hipSuccess) {
        std::cout << "The grib lons could not be copied to the GPU" << std::endl;
        hipFree(d_stationArr);
        hipFree(d_lats);
        hipFree(d_lons);
        delete[] available_gpus;
        return;
    }
    
    // call the kernel
    find_nearest_points <<< num_blocks_to_use, num_threads_to_use >>> (d_stationArr, d_lats, d_lons, numStations, numberOfPoints);
    
    // copy the elements from the GPU back over to the host
    if (hipMemcpy(stationArr, d_stationArr, hipMemcpyDeviceToHost) != hipSuccess) {
        hipFree(d_stationArr);
        hipFree(d_lats);
        hipFree(d_lons);
        delete[] available_gpus;
        return;
    }

    // release
    hipFree(d_stationArr);
    hipFree(d_lats);
    hipFree(d_lons);
    delete[] available_gpus;


}
