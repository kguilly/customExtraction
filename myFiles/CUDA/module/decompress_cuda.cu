#include "hip/hip_runtime.h"
#include "decompress_cuda.h"
#include "iostream"
#include <stdlib.h>

/*
Funtion that gets device information:
 - number of available GPUs
 - max number of threads per block
*/

deviceInfo_t device_information (int device_num) {

    hipSetDevice(device_num);
    hipDeviceProp_t gpu;
    hipGetDeviceProperties(&gpu, 0);
    deviceInfo_t dev;
    dev.name = gpu.name;
    dev.clockRate = gpu.clockRate;
    dev.maxThreadsperBlock = gpu.maxThreadsPerBlock;
    dev.maxBlocksperMulti = gpu.maxBlocksPerMultiProcessor;
    return dev;
}

/*
Function that performs the index extraction
*/
__global__ void find_nearest_points (station_t * stationArr, double * lats, double * lons, int numStations, int num_lons_lats) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < numStations) {
        double min_distance = 999;
        int min_index = -1;
        
        station_t * curr_station = &stationArr[id];
        double st_lat = (curr_station->latll + curr_station->latur) / 2.0;
        double st_lon = (curr_station->lonll + curr_station->lonur) / 2.0;

        for (int i=0; i<num_lons_lats; i++) {
            double lat = lats[i];
            double lon = lons[i]; 
            double distance = sqrt(pow((st_lat - lat), 2) + pow((st_lon - lon), 2));

            if (distance < min_distance) {
                min_distance = distance;
                min_index = i;
            }
        }
        curr_station->closestPoint = min_index;
    }
    
}

/*
Function that orchestrates the index extraction
*/
void index_extraction (station_t * stationArr, double* lats, double* lons, int numStations, int numberOfPoints) { 
    
    // get useful device properties
    int num_devices;
    hipGetDeviceCount(&num_devices);
    deviceInfo_t gpu = device_information(0);
    hipSetDevice(0);
    int max_threads = gpu.maxThreadsperBlock;
    int max_blocks = gpu.maxBlocksperMulti;

    int num_threads_to_use = max_threads / 2;
    int num_blocks_to_use = max_blocks / 2;

    // make device copies of host params
    station_t* d_stationArr;
    double* d_lats;
    double* d_lons;
    

    // allocate each of the arguments to the GPU
    if (hipMalloc(&d_stationArr, sizeof(station_t) * numStations) != hipSuccess) {
        std::cout << "stationArr could not be allocated to GPU" << std::endl;
        return;
    }
    if (hipMalloc(&d_lats, sizeof(double) * numberOfPoints) != hipSuccess) {
        std::cout << "Grib lats could not be allocated to the GPU" << std::endl;
        hipFree(d_stationArr);
        return;
    }
    if (hipMalloc(&d_lons, sizeof(double) * numberOfPoints) != hipSuccess) {
        std::cout << "Grib lons could not be allocated to the GPU" << std::endl;
        hipFree(d_stationArr);
        hipFree(d_lats);
        return;
    }

    // copy each of the arguments over to the gpu
    if (hipMemcpy(d_stationArr, stationArr, sizeof(station_t) * numStations, hipMemcpyHostToDevice) != hipSuccess) {
        std::cout << "The stationArr could not be copied to the GPU" << std::endl;
        hipFree(d_stationArr);
        hipFree(d_lats);
        hipFree(d_lons);
        return;
    }
    if (hipMemcpy(d_lats, lats, sizeof(double) * numberOfPoints, hipMemcpyHostToDevice) != hipSuccess) {
        std::cout << "The grib lats could not be copied to the GPU" << std::endl;
        hipFree(d_stationArr);
        hipFree(d_lats);
        hipFree(d_lons);
        return;
    }
    if (hipMemcpy(d_lons, lons, sizeof(double) * numberOfPoints, hipMemcpyHostToDevice) != hipSuccess) {
        std::cout << "The grib lons could not be copied to the GPU" << std::endl;
        hipFree(d_stationArr);
        hipFree(d_lats);
        hipFree(d_lons);
        return;
    }
    
    // call the kernel
    find_nearest_points <<< num_blocks_to_use, num_threads_to_use >>> (d_stationArr, d_lats, d_lons, numStations, numberOfPoints);
    hipDeviceSynchronize();

    // copy the elements from the GPU back over to the host
    if (hipMemcpy(stationArr, d_stationArr, sizeof(station_t) * numStations, hipMemcpyDeviceToHost) != hipSuccess) {
        hipFree(d_stationArr);
        hipFree(d_lats);
        hipFree(d_lons);
        return;
    }
    std::cout << "cuda success" << std::endl;
    // release
    hipFree(d_stationArr);
    hipFree(d_lats);
    hipFree(d_lons);

}
