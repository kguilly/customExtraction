#include "hip/hip_runtime.h"
#include "test_kernel.h"
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <ctime>
#include <hip/hip_runtime.h>

__global__
void add_ints(int* a, int* b, int count){
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < count) {
    a[id] += b[id];
  }
}

void the_kernel(int* x, int* y, int N){
  // Make device versions of the host variables
  int *d_x, *d_y;
  if (hipMalloc(&d_x, sizeof(int)*N) != hipSuccess){
  std::cout << "d_x could not be alloced" << std::endl;
  return;
  }

  if (hipMalloc(&d_y, sizeof(int)*N) != hipSuccess){
  std::cout << "d_y could not be alloced" << std::endl;
  hipFree(d_x);
  return;
  }

  // Now copy the values over to the GPU
  if (hipMemcpy(d_x, x, sizeof(int) * N, hipMemcpyHostToDevice) != hipSuccess){
  std::cout << "Could not copy d_x" << std::endl;
  hipFree(d_x);
  hipFree(d_y);
  return;
  }

  if (hipMemcpy(d_y, y, sizeof(int) * N, hipMemcpyHostToDevice) != hipSuccess){
  std::cout << "Could not copy d_y" << std::endl;
  hipFree(d_x);
  hipFree(d_y);
  return;
  }

  add_ints<<<N / 256 + 1, 256 >>>(d_x, d_y, N);

  if (hipMemcpy(x, d_x, sizeof(int) * N, hipMemcpyDeviceToHost) != hipSuccess) {
      delete[] x;
      delete[] y;  
      hipFree(d_x);
      hipFree(d_y);
      std::cout << "The stuff could not be copied back to the host" << std::endl;
      return;
  }

  hipFree(d_x);
  hipFree(d_y);
}