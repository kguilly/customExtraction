#include "hip/hip_runtime.h"
// this test will add the elements of two arrays with a million elemetns each
// I also want to test how to get information about the GPU
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays

__global__
void add(int, float*, float*);

int main(void)
{

  int N = 1<<20; // I think thats a binary 1 shifted left 20 times
  float *x, *y;

  /////////////////////////////////////////////////////////////////////
  // Get device properties

  int num_devices;
  hipGetDeviceCount(&num_devices);
  for(int i=0; i < num_devices; i++){
    hipDeviceProp_t prop;
    cudaGetDeviceProperties_v2(&prop, i);
    std::cout << "\nDevice " << i << ": " << prop.name << "\n - Clock Rate: ";
    std::cout << prop.clockRate << "\n - Compute Mode: " << prop.computeMode;
    std::cout << "\n - MaxGridSize: " << prop.maxGridSize << "\n - Max Threads Per Block: ";
    std::cout << prop.maxThreadsPerBlock << "\n - Max Threads Dim: " << prop.maxThreadsDim << std::endl;
    std::cout << " - UUID: " << /*prop.uuid*/ "\n - Warp Size: " << prop.warpSize << std::endl;
    std::cout << " - Max Blocks per multiprocessor: " << prop.maxBlocksPerMultiProcessor << std::endl;
    
    size_t free_mem, total_mem;
    hipSetDevice(i);
    hipMemGetInfo(&free_mem, &total_mem);
    std::cout << "Total Memory: " << total_mem << "\t Free Memory: " << free_mem << std::endl;

  }
  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // make a nice block size
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  // Run kernel on 1M elements on the GPU
  add<<<numBlocks, blockSize>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}

__global__
void add(int n, float *x, float *y){
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
  
}